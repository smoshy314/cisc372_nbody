#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include <hip/hip_runtime.h>

__global__ void accelComputeKernal(vector3* dev_accels, double * dev_mass, vector3* dev_hPos){
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int index = i * NUMENTITIES + j;

	if (i < NUMENTITIES && j < NUMENTITIES) {
		if (i==j) {
			FILL_VECTOR(dev_accels[index],0,0,0);
		}else{
			vector3 distance;
			for (int k=0;k<3;k++) distance[k]=dev_hPos[i][k]-dev_hPos[j][k];
			double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
			double magnitude=sqrt(magnitude_sq);
			double accelmag=-1*GRAV_CONSTANT*dev_mass[j]/magnitude_sq;
			FILL_VECTOR(dev_accels[index],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);	
		}
	}
}

// __global__ void contructAccels(vector3** dev_accels, vector3* dev_values){
// 	int i = blockIdx.x * blockDim.x + threadIdx.x;
// 	if (i < NUMENTITIES){
// 		dev_accels[i] = &dev_values[i*NUMENTITIES];
// 	}
// }

__global__ void sumRows(vector3* dev_accels, vector3* dev_hPos, vector3* dev_hVel){
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int k;

	if(i < NUMENTITIES && j< NUMENTITIES){
	vector3 accel_sum={0,0,0};
	for (k=0;k<3;k++)
		accel_sum[k]+=dev_accels[i * NUMENTITIES + j][k];

		
//compute the new velocity based on the acceleration and time interval
//compute the new position based on the velocity and time interval
	for (k=0;k<3;k++){	
		dev_hVel[i][k]+=accel_sum[k]*INTERVAL;
		dev_hPos[i][k]+=dev_hVel[i][k]*INTERVAL;
	}
}
}
//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
void compute(){
	//make an acceleration matrix which is NUMENTITIES squared in size;
	int i,j,k;
	//vector3* dev_values;
	// vector3* values=(vector3*)malloc(sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	// hipMalloc(&dev_values, sizeof(vector3) * NUMENTITIES * NUMENTITIES);
	// hipMemcpy(dev_values, values,sizeof(vector3) * NUMENTITIES * NUMENTITIES,hipMemcpyHostToDevice);
	// vector3** dev_accels;
	// hipMalloc(&dev_accels, sizeof(vector3*) * NUMENTITIES);
	
	vector3* dev_accels;
	hipMalloc(&dev_accels, sizeof(vector3) * NUMENTITIES * NUMENTITIES);

	

	// contructAccels<<<dimGrid, dimAc>>>(dev_accels, dev_values);

	double * dev_mass;
	vector3* dev_hPos;
	vector3* dev_hVel;
	hipMalloc(&dev_mass, sizeof(double) * NUMENTITIES );
	hipMemcpy(dev_mass, mass, sizeof(double) * NUMENTITIES,hipMemcpyHostToDevice);
	hipMalloc(&dev_hPos, sizeof(vector3) * NUMENTITIES );
	hipMemcpy(dev_hPos, hPos,sizeof(vector3) * NUMENTITIES,hipMemcpyHostToDevice);
	hipMalloc(&dev_hVel, sizeof(vector3) * NUMENTITIES );
	hipMemcpy(dev_hVel, hVel,sizeof(vector3) * NUMENTITIES,hipMemcpyHostToDevice);
	
	//dim3 numBlocks((NUMENTITIES+15)/16, (NUMENTITIES+15)/16);
	dim3 blockSize(32, 32);
	int blockDimX = 32;
	int blockDimY = 32;

	int gridDim = (NUMENTITIES + blockDimX - 1) / blockDimX; 
	for (t_now=0;t_now<DURATION;t_now+=INTERVAL){
		accelComputeKernal<<<gridDim, blockSize>>>(dev_accels, dev_mass, dev_hPos);
		hipDeviceSynchronize();
		sumRows<<<gridDim, blockSize>>>(dev_accels, dev_hPos, dev_hVel);
		hipError_t hipError_t = hipGetLastError();
		if (hipError_t != hipSuccess) {
			printf("CUDA Error: %s\n", hipGetErrorString(hipError_t));
		}
	}
	hipMemcpy(hVel, dev_hVel, sizeof(vector3)*NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(hPos, dev_hPos, sizeof(vector3)*NUMENTITIES, hipMemcpyDeviceToHost);
	//sum up the rows of our matrix to get effect on each entity, then update velocity and position.
	
	//free(accels);
	hipFree(dev_mass);
	hipFree(dev_accels);
	hipFree(dev_hPos);
	hipFree(dev_hVel);
}
