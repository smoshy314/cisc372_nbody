#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include <hip/hip_runtime.h>

__global__ void accelComputeKernal(vector3* dev_accels, double * dev_mass, vector3* dev_hPos){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int index = i * NUMENTITIES + j;

	if (i < NUMENTITIES && j < NUMENTITIES) {
		if (i==j) {
			FILL_VECTOR(dev_accels[index],0,0,0);
		}else{
			vector3 distance;
			for (k=0;k<3;k++) distance[k]=hPos[i][k]-hPos[j][k];
			double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
			double magnitude=sqrt(magnitude_sq);
			double accelmag=-1*GRAV_CONSTANT*mass[j]/magnitude_sq;
			FILL_VECTOR(accels[index],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);	
		}
	}
}

// __global__ void contructAccels(vector3** dev_accels, vector3* dev_values){
// 	int i = blockIdx.x * blockDim.x + threadIdx.x;
// 	if (i < NUMENTITIES){
// 		dev_accels[i] = &dev_values[i*NUMENTITIES];
// 	}
// }

__global__ void sumRows(vector3* dev_accels, vector3* dev_hPos, vector3* dev_hVel){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k = threadIdx.z;

	if(i < NUMENTITIES && j< NUMENTITIES){
	vector3 accel_sum={0,0,0};
	accel_sum[k]+=dev_accels[i * NUMENTITIES + j][k];
		
//compute the new velocity based on the acceleration and time interval
//compute the new position based on the velocity and time interval
	dev_hVel[i][k]+=accel_sum[k]*INTERVAL;
	dev_hPos[i][k]+=dev_hVel[i][k]*INTERVAL;
	}
}
//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
void compute(){
	//make an acceleration matrix which is NUMENTITIES squared in size;
	int i,j,k;
	//vector3* dev_values;
	// vector3* values=(vector3*)malloc(sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	// hipMalloc(&dev_values, sizeof(vector3) * NUMENTITIES * NUMENTITIES);
	// hipMemcpy(dev_values, values,sizeof(vector3) * NUMENTITIES * NUMENTITIES,hipMemcpyHostToDevice);

	int gridD = (NUMENTITIES/256) +1;
	// vector3** dev_accels;
	// hipMalloc(&dev_accels, sizeof(vector3*) * NUMENTITIES);
	vector3* dev_accels;
	hipMalloc(&dev_accels, sizeof(vector3) * NUMENTITIES * NUMENTITIES);

	

	// contructAccels<<<dimGrid, dimAc>>>(dev_accels, dev_values);

	double * dev_mass;
	vector3* dev_hPos;
	vector3* dev_hVel;
	hipMalloc(&dev_mass, sizeof(double) * NUMENTITIES );
	hipMemcpy(dev_mass, mass, sizeof(double) * NUMENTITIES,hipMemcpyHostToDevice);
	hipMalloc(&dev_hPos, sizeof(vector3) * NUMENTITIES );
	hipMemcpy(dev_hPos, hPos,sizeof(vector3) * NUMENTITIES,hipMemcpyHostToDevice);
	hipMalloc(&dev_hVel, sizeof(vector3) * NUMENTITIES );
	hipMemcpy(dev_hVel, hVel,sizeof(vector3) * NUMENTITIES,hipMemcpyHostToDevice);
	
	//dim3 numBlocks((NUMENTITIES+15)/16, (NUMENTITIES+15)/16);
	dim3 blockSize(32, 32);
	int blockDimX = 32;
	int blockDimY = 32;

	int gridDim = (NUMENTITIES + blockDimX - 1) / blockDimX; 
	dim3 grid(gridDim, gridDim);
	accelComputeKernal<<<gridDim, blockSize>>>(dev_accels, dev_mass, dev_hPos);
	hipDeviceSynchronize();
	// vector3* accels = (vector3*)malloc(sizeof(vector3) * NUMENTITIES * NUMENTITIES);
	// hipMemcpy(accels, dev_accels, sizeof(vector3)*NUMENTITIES, hipMemcpyDeviceToHost);
	// for (int i = 0; i < NUMENTITIES; ++i) {
	// 	for (int j = 0; j < NUMENTITIES; ++j) {
	// 		printf("(%f, %f, %f) ", accels[i * NUMENTITIES + j][0], accels[i * NUMENTITIES + j][1], accels[i * NUMENTITIES + j][2]);
	// 	}
	// 	printf("\n");
	// }
	dim3 sblockSize(16, 16, 3);
	int sblockDimX = 16;
	int sblockDimY = 16;

	int sgridDim = (NUMENTITIES + sblockDimX - 1) / sblockDimX; 	
	sumRows<<<sgridDim, sblockSize>>>(dev_accels, dev_hPos, dev_hVel);
	hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(hipError_t));
	}
	hipMemcpy(hVel, dev_hVel, sizeof(vector3)*NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(hPos, dev_hPos, sizeof(vector3)*NUMENTITIES, hipMemcpyDeviceToHost);
	//sum up the rows of our matrix to get effect on each entity, then update velocity and position.
	
	//free(accels);
	hipFree(dev_mass);
	hipFree(dev_accels);
	hipFree(dev_hPos);
	hipFree(dev_hVel);
}
